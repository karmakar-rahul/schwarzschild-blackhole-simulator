#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>

#define WIDTH 1600
#define HEIGHT 900
#define MAX_STEPS 2000
#define STEP_SIZE 0.006f
#define SCHWARZSCHILD_RADIUS 3.0f
#define TEXTURE_WIDTH 4096
#define TEXTURE_HEIGHT 4096

// Camera state
struct Camera {
    float distance;
    float theta;
    float phi;
    float rotation_speed;
};

Camera g_camera = {15.0f, 1.2f, 0.0f, 0.4f};
bool g_mouse_pressed = false;
double g_last_x = 0.0, g_last_y = 0.0;

// Vector operations
struct Vec3 {
    float x, y, z;
};

__device__ Vec3 vec3_add(Vec3 a, Vec3 b) {
    return {a.x + b.x, a.y + b.y, a.z + b.z};
}

__device__ Vec3 vec3_sub(Vec3 a, Vec3 b) {
    return {a.x - b.x, a.y - b.y, a.z - b.z};
}

__device__ Vec3 vec3_scale(Vec3 v, float s) {
    return {v.x * s, v.y * s, v.z * s};
}

__device__ float vec3_dot(Vec3 a, Vec3 b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ float vec3_length(Vec3 v) {
    return sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ Vec3 vec3_normalize(Vec3 v) {
    float len = vec3_length(v);
    if (len > 1e-6f) {
        return {v.x / len, v.y / len, v.z / len};
    }
    return {0.0f, 0.0f, 1.0f};
}

// Schwarzschild acceleration - STRONGER lensing
__device__ Vec3 compute_acceleration(Vec3 pos, Vec3 vel) {
    float r = vec3_length(pos);
    
    if (r < SCHWARZSCHILD_RADIUS * 1.01f) {
        return {0.0f, 0.0f, 0.0f};
    }
    
    float rs = SCHWARZSCHILD_RADIUS;
    float r2 = r * r;
    float r3 = r2 * r;
    
    Vec3 r_hat = vec3_normalize(pos);
    float v_r = vec3_dot(vel, r_hat);
    
    Vec3 v_tang = vec3_sub(vel, vec3_scale(r_hat, v_r));
    float v_tang2 = vec3_dot(v_tang, v_tang);
    
    // Increase lensing strength by 2.5x
    float factor = -3.75f * rs / r3;
    Vec3 accel = vec3_scale(pos, factor * v_tang2);
    
    return accel;
}

// Sample from background texture using spherical mapping with bilinear filtering
__device__ Vec3 sample_background_texture(Vec3 dir, unsigned char* texture, int tex_width, int tex_height) {
    // Convert direction to spherical coordinates
    float theta = atan2f(dir.z, dir.x);
    float phi = asinf(fmaxf(-1.0f, fminf(1.0f, dir.y))); // Clamp to avoid NaN
    
    // Map to texture coordinates [0, 1]
    float u = (theta + 3.14159265f) / (2.0f * 3.14159265f);
    float v = (phi + 3.14159265f / 2.0f) / 3.14159265f;
    
    // Wrap coordinates
    u = u - floorf(u);
    v = v - floorf(v);
    
    // Get float coordinates
    float fx = u * (tex_width - 1);
    float fy = v * (tex_height - 1);
    
    // Bilinear interpolation
    int x0 = (int)fx;
    int y0 = (int)fy;
    int x1 = (x0 + 1) % tex_width;
    int y1 = (y0 + 1) % tex_height;
    
    float wx = fx - x0;
    float wy = fy - y0;
    
    // Sample 4 neighboring pixels
    int idx00 = (y0 * tex_width + x0) * 3;
    int idx10 = (y0 * tex_width + x1) * 3;
    int idx01 = (y1 * tex_width + x0) * 3;
    int idx11 = (y1 * tex_width + x1) * 3;
    
    Vec3 color;
    // Interpolate red channel
    float r00 = texture[idx00 + 0] / 255.0f;
    float r10 = texture[idx10 + 0] / 255.0f;
    float r01 = texture[idx01 + 0] / 255.0f;
    float r11 = texture[idx11 + 0] / 255.0f;
    color.x = (1.0f - wx) * (1.0f - wy) * r00 + wx * (1.0f - wy) * r10 +
              (1.0f - wx) * wy * r01 + wx * wy * r11;
    
    // Interpolate green channel
    float g00 = texture[idx00 + 1] / 255.0f;
    float g10 = texture[idx10 + 1] / 255.0f;
    float g01 = texture[idx01 + 1] / 255.0f;
    float g11 = texture[idx11 + 1] / 255.0f;
    color.y = (1.0f - wx) * (1.0f - wy) * g00 + wx * (1.0f - wy) * g10 +
              (1.0f - wx) * wy * g01 + wx * wy * g11;
    
    // Interpolate blue channel
    float b00 = texture[idx00 + 2] / 255.0f;
    float b10 = texture[idx10 + 2] / 255.0f;
    float b01 = texture[idx01 + 2] / 255.0f;
    float b11 = texture[idx11 + 2] / 255.0f;
    color.z = (1.0f - wx) * (1.0f - wy) * b00 + wx * (1.0f - wy) * b10 +
              (1.0f - wx) * wy * b01 + wx * wy * b11;
    
    return color;
}

// Background with procedural galaxy and stars (fallback if no texture)
__device__ Vec3 get_background_color(Vec3 dir, unsigned char* texture, int tex_width, int tex_height) {
    // If texture is available, use it
    if (texture != nullptr) {
        Vec3 tex_color = sample_background_texture(dir, texture, tex_width, tex_height);
        
        // Add some extra stars on top
        float star_density = 600.0f;
        float star = sinf(dir.x * star_density) * cosf(dir.y * star_density) * sinf(dir.z * star_density);
        star = star * star * star * star;
        star = fmaxf(0.0f, star - 0.996f) * 400.0f;
        
        tex_color.x += star * 0.3f;
        tex_color.y += star * 0.3f;
        tex_color.z += star * 0.3f;
        
        return tex_color;
    }
    
    // Fallback: procedural background
    // Stars - brighter and more visible
    float star_density = 500.0f;
    float star = sinf(dir.x * star_density) * cosf(dir.y * star_density) * sinf(dir.z * star_density);
    star = star * star * star * star;
    star = fmaxf(0.0f, star - 0.993f) * 300.0f;
    
    // Dark space background
    Vec3 bg = {0.01f + star, 0.01f + star, 0.02f + star};
    
    // Bright orange/red nebula
    float nebula_scale = 1.5f;
    float nebula = sinf(dir.x * nebula_scale) * cosf(dir.y * nebula_scale * 1.3f) * sinf(dir.z * nebula_scale * 0.7f);
    nebula = nebula * 0.5f + 0.5f;
    nebula = powf(nebula, 2.0f) * 0.25f;
    
    bg.x += nebula * 1.2f;  // Strong red
    bg.y += nebula * 0.4f;  // Medium orange
    bg.z += nebula * 0.1f;  // Minimal blue
    
    // Accretion disk glow in background (horizontal band)
    float disk_angle = fabsf(dir.y);
    if (disk_angle < 0.5f) {
        float disk_intensity = (0.5f - disk_angle) / 0.5f;
        
        // Multi-layer disk for more realism
        float inner_disk = powf(disk_intensity, 0.8f) * 1.2f;
        float outer_disk = powf(disk_intensity, 2.0f) * 0.8f;
        
        float total_intensity = inner_disk + outer_disk;
        
        // Hot plasma colors
        bg.x += total_intensity * 1.0f;   // Red
        bg.y += total_intensity * 0.6f;   // Orange/yellow
        bg.z += total_intensity * 0.1f;   // Slight blue
    }
    
    return bg;
}

// Ray tracing (grid removed)
__device__ Vec3 trace_ray(Vec3 origin, Vec3 direction, unsigned char* bg_texture, int tex_width, int tex_height) {
    Vec3 pos = origin;
    Vec3 vel = vec3_normalize(direction);
    
    // Check if ray starts inside event horizon
    float start_r = vec3_length(pos);
    if (start_r < SCHWARZSCHILD_RADIUS * 2.0f) {
        return get_background_color(direction, bg_texture, tex_width, tex_height);
    }
    
    for (int step = 0; step < MAX_STEPS; step++) {
        float r = vec3_length(pos);
        
        // Hit event horizon - return black
        if (r < SCHWARZSCHILD_RADIUS * 1.5f) {
            return {0.0f, 0.0f, 0.0f};
        }
        
        // Escaped to infinity
        if (r > 50.0f) {
            return get_background_color(vel, bg_texture, tex_width, tex_height);
        }
        
        // Apply gravitational deflection
        Vec3 accel = compute_acceleration(pos, vel);
        vel = vec3_add(vel, vec3_scale(accel, STEP_SIZE));
        vel = vec3_normalize(vel);
        pos = vec3_add(pos, vec3_scale(vel, STEP_SIZE));
    }
    
    // Max steps reached - show background
    return get_background_color(vel, bg_texture, tex_width, tex_height);
}

// CUDA kernel - renders to simple buffer
__global__ void render_kernel(unsigned char* image, int width, int height, 
                              float cam_x, float cam_y, float cam_z,
                              float right_x, float right_y, float right_z,
                              float up_x, float up_y, float up_z,
                              float forward_x, float forward_y, float forward_z,
                              unsigned char* bg_texture, int tex_width, int tex_height) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (px >= width || py >= height) return;
    
    Vec3 camera_pos = {cam_x, cam_y, cam_z};
    Vec3 right = {right_x, right_y, right_z};
    Vec3 up = {up_x, up_y, up_z};
    Vec3 forward = {forward_x, forward_y, forward_z};
    
    float u = (2.0f * px / width - 1.0f) * (float)width / height;
    float v = 1.0f - 2.0f * py / height;
    
    Vec3 ray_dir = vec3_normalize(vec3_add(
        vec3_add(vec3_scale(right, u * 0.8f), vec3_scale(up, v * 0.8f)),
        forward
    ));
    
    Vec3 color = trace_ray(camera_pos, ray_dir, bg_texture, tex_width, tex_height);
    
    // Tone mapping and gamma
    color.x = fminf(1.0f, color.x);
    color.y = fminf(1.0f, color.y);
    color.z = fminf(1.0f, color.z);
    
    color.x = powf(color.x, 1.0f / 2.2f);
    color.y = powf(color.y, 1.0f / 2.2f);
    color.z = powf(color.z, 1.0f / 2.2f);
    
    // Write RGB (flipped for OpenGL)
    int idx = ((height - 1 - py) * width + px) * 3;
    image[idx + 0] = (unsigned char)(color.x * 255.0f);
    image[idx + 1] = (unsigned char)(color.y * 255.0f);
    image[idx + 2] = (unsigned char)(color.z * 255.0f);
}

// Mouse callbacks
void mouse_button_callback(GLFWwindow* window, int button, int action, int mods) {
    if (button == GLFW_MOUSE_BUTTON_LEFT) {
        if (action == GLFW_PRESS) {
            g_mouse_pressed = true;
            glfwGetCursorPos(window, &g_last_x, &g_last_y);
        } else if (action == GLFW_RELEASE) {
            g_mouse_pressed = false;
        }
    }
}

void cursor_position_callback(GLFWwindow* window, double xpos, double ypos) {
    if (g_mouse_pressed) {
        double dx = xpos - g_last_x;
        double dy = ypos - g_last_y;
        
        g_camera.phi += dx * 0.005f;
        g_camera.theta += dy * 0.005f;
        
        if (g_camera.theta < 0.1f) g_camera.theta = 0.1f;
        if (g_camera.theta > 3.04f) g_camera.theta = 3.04f;
        
        // Pause auto-rotation when user interacts
        g_camera.rotation_speed = 0.0f;
        
        g_last_x = xpos;
        g_last_y = ypos;
    }
}

void scroll_callback(GLFWwindow* window, double xoffset, double yoffset) {
    g_camera.distance -= yoffset * 0.5f;
    if (g_camera.distance < 5.0f) g_camera.distance = 5.0f;
    if (g_camera.distance > 30.0f) g_camera.distance = 30.0f;
}

void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS) {
        glfwSetWindowShouldClose(window, GLFW_TRUE);
    }
    if (key == GLFW_KEY_R && action == GLFW_PRESS) {
        g_camera = {15.0f, 1.2f, 0.0f, 0.15f};
    }
    if (key == GLFW_KEY_SPACE && action == GLFW_PRESS) {
        // Toggle auto-rotation
        if (g_camera.rotation_speed == 0.0f) {
            g_camera.rotation_speed = 0.4f;
        } else {
            g_camera.rotation_speed = 0.0f;
        }
    }
}

// Load image function (simple PPM loader)
unsigned char* load_ppm_image(const char* filename, int* width, int* height) {
    FILE* fp = fopen(filename, "rb");
    if (!fp) {
        printf("Warning: Could not open background image '%s'\n", filename);
        printf("Using procedural background instead.\n");
        return nullptr;
    }
    
    char buffer[16];
    int max_val;
    
    // Read PPM header
    if (!fgets(buffer, sizeof(buffer), fp)) {
        fclose(fp);
        return nullptr;
    }
    
    // Check format (P6 = binary PPM)
    if (buffer[0] != 'P' || buffer[1] != '6') {
        printf("Warning: Image must be in PPM P6 format\n");
        fclose(fp);
        return nullptr;
    }
    
    // Skip comments
    do {
        if (!fgets(buffer, sizeof(buffer), fp)) {
            fclose(fp);
            return nullptr;
        }
    } while (buffer[0] == '#');
    
    // Read dimensions
    sscanf(buffer, "%d %d", width, height);
    
    // Read max value
    if (!fgets(buffer, sizeof(buffer), fp)) {
        fclose(fp);
        return nullptr;
    }
    sscanf(buffer, "%d", &max_val);
    
    // Allocate and read image data
    size_t image_size = (*width) * (*height) * 3;
    unsigned char* data = (unsigned char*)malloc(image_size);
    
    if (fread(data, 1, image_size, fp) != image_size) {
        printf("Warning: Could not read full image\n");
        free(data);
        fclose(fp);
        return nullptr;
    }
    
    fclose(fp);
    printf("Loaded background image: %dx%d\n", *width, *height);
    return data;
}

int main() {
    printf("Interactive Schwarzschild Black Hole Simulator\n");
    printf("Controls:\n");
    printf("  - Left Click + Drag: Manual camera control (pauses auto-rotation)\n");
    printf("  - Mouse Wheel: Zoom in/out\n");
    printf("  - SPACE: Toggle auto-rotation on/off\n");
    printf("  - R: Reset camera\n");
    printf("  - ESC: Exit\n\n");
    
    // Load background texture
    int bg_width = 0, bg_height = 0;
    unsigned char* h_bg_texture = load_ppm_image("galaxy_background.ppm", &bg_width, &bg_height);
    
    // Upload background texture to GPU
    unsigned char* d_bg_texture = nullptr;
    if (h_bg_texture) {
        size_t bg_size = bg_width * bg_height * 3;
        hipMalloc(&d_bg_texture, bg_size);
        hipMemcpy(d_bg_texture, h_bg_texture, bg_size, hipMemcpyHostToDevice);
        free(h_bg_texture);
    } else {
        printf("Note: Place a 'galaxy_background.ppm' file in the same directory for custom background\n");
        printf("You can convert any image to PPM using: convert image.jpg -resize 4096x4096! galaxy_background.ppm\n\n");
    }
    
    // Check CUDA
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA devices found!\n");
        return -1;
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Using GPU: %s\n", prop.name);
    
    // Initialize GLFW
    if (!glfwInit()) {
        fprintf(stderr, "Failed to initialize GLFW\n");
        return -1;
    }
    
    // Create window
    GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "Black Hole Simulator", NULL, NULL);
    if (!window) {
        fprintf(stderr, "Failed to create window\n");
        glfwTerminate();
        return -1;
    }
    
    glfwMakeContextCurrent(window);
    glfwSetMouseButtonCallback(window, mouse_button_callback);
    glfwSetCursorPosCallback(window, cursor_position_callback);
    glfwSetScrollCallback(window, scroll_callback);
    glfwSetKeyCallback(window, key_callback);
    glfwSwapInterval(0); // Disable V-sync for higher FPS
    
    // Initialize GLEW
    if (glewInit() != GLEW_OK) {
        fprintf(stderr, "Failed to initialize GLEW\n");
        return -1;
    }
    
    // Create OpenGL texture (no CUDA interop)
    GLuint texture;
    glGenTextures(1, &texture);
    glBindTexture(GL_TEXTURE_2D, texture);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    
    // Allocate buffers
    size_t image_size = WIDTH * HEIGHT * 3;
    unsigned char* h_image = (unsigned char*)malloc(image_size);
    unsigned char* d_image;
    hipMalloc(&d_image, image_size);
    
    // CUDA kernel config
    dim3 block_size(16, 16);
    dim3 grid_size((WIDTH + block_size.x - 1) / block_size.x,
                   (HEIGHT + block_size.y - 1) / block_size.y);
    
    printf("Rendering at %dx%d...\n\n", WIDTH, HEIGHT);
    
    int frame_count = 0;
    double last_time = glfwGetTime();
    double fps_time = last_time;
    int fps_frames = 0;
    
    // Main loop
    while (!glfwWindowShouldClose(window)) {
        // Calculate delta time for smooth rotation
        double current_time = glfwGetTime();
        double delta_time = current_time - last_time;
        last_time = current_time;
        
        // Auto-rotate camera with frame-independent speed
        g_camera.phi += g_camera.rotation_speed * delta_time;
        
        // Calculate camera
        float cam_x = g_camera.distance * sinf(g_camera.theta) * cosf(g_camera.phi);
        float cam_y = g_camera.distance * cosf(g_camera.theta);
        float cam_z = g_camera.distance * sinf(g_camera.theta) * sinf(g_camera.phi);
        
        float forward_x = -cam_x / g_camera.distance;
        float forward_y = -cam_y / g_camera.distance;
        float forward_z = -cam_z / g_camera.distance;
        
        float right_x = -sinf(g_camera.phi);
        float right_y = 0.0f;
        float right_z = cosf(g_camera.phi);
        
        float up_x = -cosf(g_camera.theta) * cosf(g_camera.phi);
        float up_y = sinf(g_camera.theta);
        float up_z = -cosf(g_camera.theta) * sinf(g_camera.phi);
        
        // Render with CUDA
        render_kernel<<<grid_size, block_size>>>(
            d_image, WIDTH, HEIGHT,
            cam_x, cam_y, cam_z,
            right_x, right_y, right_z,
            up_x, up_y, up_z,
            forward_x, forward_y, forward_z,
            d_bg_texture, bg_width, bg_height
        );
        
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(err));
            break;
        }
        
        hipDeviceSynchronize();
        
        // Copy to host
        hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost);
        
        // Upload to OpenGL
        glBindTexture(GL_TEXTURE_2D, texture);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, WIDTH, HEIGHT, 0, GL_RGB, GL_UNSIGNED_BYTE, h_image);
        
        // Render quad
        glClear(GL_COLOR_BUFFER_BIT);
        glEnable(GL_TEXTURE_2D);
        
        glBegin(GL_QUADS);
        glTexCoord2f(0, 0); glVertex2f(-1, -1);
        glTexCoord2f(1, 0); glVertex2f(1, -1);
        glTexCoord2f(1, 1); glVertex2f(1, 1);
        glTexCoord2f(0, 1); glVertex2f(-1, 1);
        glEnd();
        
        glfwSwapBuffers(window);
        glfwPollEvents();
        
        frame_count++;
        fps_frames++;
        
        // Display FPS every second
        if (current_time - fps_time >= 1.0) {
            double fps = fps_frames / (current_time - fps_time);
            printf("FPS: %.1f | Frames: %d\r", fps, frame_count);
            fflush(stdout);
            fps_frames = 0;
            fps_time = current_time;
        }
    }
    
    printf("\nExiting...\n");
    
    // Cleanup
    if (d_bg_texture) hipFree(d_bg_texture);
    hipFree(d_image);
    free(h_image);
    glDeleteTextures(1, &texture);
    glfwDestroyWindow(window);
    glfwTerminate();
    
    return 0;
}

